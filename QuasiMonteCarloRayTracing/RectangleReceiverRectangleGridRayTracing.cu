#include "hip/hip_runtime.h"
#include "RectangleReceiverRectangleGridRayTracing.cuh"
#include "Receiver/rectangleReceiverIntersection.cuh"
#include "Grid/rectGridDDA.cuh"


__global__ void map_raytracing(SunrayArgument sunrayArgument, RectangleReceiver rectangleReceiver,
                                RectangleGrid rectangleGrid, HeliostatArgument heliostatArgument,
                               float3 *d_subheliostat_vertexes, float factor){
    long long myId = global_func::getThreadID();
    if(myId >= heliostatArgument.numberOfMicroHeliostats * sunrayArgument.numberOfLightsPerGroup){
        return;
    }

    // Step 1: whether the incident light is shadowed by other heliostats.
    int address = ( heliostatArgument.d_microHelio_groups[myId / sunrayArgument.numberOfLightsPerGroup] +
            myId % sunrayArgument.numberOfLightsPerGroup ) % sunrayArgument.pool_size;
    float3 dir = global_func::local2world_rotate(sunrayArgument.d_samplelights[address], -sunrayArgument.sunray_direction);
    float3 origin = heliostatArgument.d_microHelio_origins[myId / sunrayArgument.numberOfLightsPerGroup];

    if(rectGridDDA::collision(origin, dir, rectangleGrid, d_subheliostat_vertexes, heliostatArgument)){
        return;
    }

    // Step 2: whether the reflect light is shadowed by other heliostats.
    float3 normal = heliostatArgument.d_microHelio_normals[myId / sunrayArgument.numberOfLightsPerGroup];
    address = (heliostatArgument.d_microHelio_groups[(myId / sunrayArgument.numberOfLightsPerGroup + 1) %
                                                     sunrayArgument.pool_size] +
                    myId % sunrayArgument.numberOfLightsPerGroup) % sunrayArgument.pool_size;
    normal = global_func::local2world_rotate(sunrayArgument.d_perturbations[address], normal);
    normal = normalize(normal);

    dir = normalize(reflect(-dir, normal));

    if(rectGridDDA::collision(origin, dir, rectangleGrid, d_subheliostat_vertexes, heliostatArgument)){
        return;
    }

    // Step 3: intersect with receiver and calculate energy.
    rectangleReceiverIntersect::receiver_drawing(rectangleReceiver, origin, dir, normal, factor);

}



void RectangleReceiverRectangleGridRayTracing(SunrayArgument &sunrayArgument, RectangleReceiver *rectangleReceiver,
                                         RectangleGrid *rectGrid, HeliostatArgument &heliostatArgument,
                                         float3 *d_subHeliostat_vertexes, float factor) {
    int nThreads = 512;
    dim3 nBlocks;
    global_func::setThreadBlocks(nBlocks, nThreads,
                                  heliostatArgument.numberOfMicroHeliostats * sunrayArgument.numberOfLightsPerGroup,
                                  true);
    map_raytracing << < nBlocks, nThreads >> >
                                 (sunrayArgument, *rectangleReceiver, *rectGrid, heliostatArgument, d_subHeliostat_vertexes, factor);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}