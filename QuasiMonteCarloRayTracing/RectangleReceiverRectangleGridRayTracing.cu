#include "hip/hip_runtime.h"
#include "RectangleReceiverRectangleGridRayTracing.cuh"
#include "Receiver/rectangleReceiverIntersection.cuh"
#include "Grid/rectGridDDA.cuh"


__global__ void map_raytracing(SunrayArgument sunrayArgument, RectangleReceiver rectangleReceiver,
                                RectangleGrid rectangleGrid, HeliostatArgument heliostatArgument,
                               float3 *d_subheliostat_vertexes, float factor){
    long long myId = global_func::getThreadID();
    if(myId >= heliostatArgument.numberOfMicroHeliostats * sunrayArgument.numberOfLightsPerGroup){
        return;
    }

    // Step 1: whether the incident light is shadowed by other heliostats.  判断入射光
    int address = ( heliostatArgument.d_microHelio_groups[myId / sunrayArgument.numberOfLightsPerGroup] + myId % sunrayArgument.numberOfLightsPerGroup )
                        % sunrayArgument.pool_size;
    float3 dir = global_func::local2world_rotate(sunrayArgument.d_samplelights[address], -sunrayArgument.sunray_direction);
    float3 origin = heliostatArgument.d_microHelio_origins[myId / sunrayArgument.numberOfLightsPerGroup];

    if(rectGridDDA::collision(origin, dir, rectangleGrid, d_subheliostat_vertexes, heliostatArgument)){
        return;
    }

    // Step 2: whether the reflect light is shadowed by other heliostats. 判断反射光
    float3 normal = heliostatArgument.d_microHelio_normals[myId / sunrayArgument.numberOfLightsPerGroup];
    address = (heliostatArgument.d_microHelio_groups[(myId / sunrayArgument.numberOfLightsPerGroup + 1) % sunrayArgument.pool_size] + myId % sunrayArgument.numberOfLightsPerGroup)
                    % sunrayArgument.pool_size;
    normal = global_func::local2world_rotate(sunrayArgument.d_perturbations[address], normal);
    normal = normalize(normal);

    dir = normalize(reflect(-dir, origin));

    if(rectGridDDA::collision(origin, dir, rectangleGrid, d_subheliostat_vertexes, heliostatArgument)){
        return;
    }

    // Step 3: intersect with receiver. 和接收器求交，计算能量
    rectangleReceiverIntersect::receiver_drawing(rectangleReceiver, origin, dir, normal, factor);

}



void RectangleReceiverRectangleGridRayTracing( SunrayArgument &sunrayArgument,
                                                RectangleReceiver *rectangleReceiver,
                                                RectangleGrid *rectangleGrid,
                                                HeliostatArgument *heliostatArgument,
                                                float3 *d_subHeliostat_vertexes,
                                                float factor ){
    int nThreads = 512;
    dim3 nBlocks;
    global_func::setThreadBlocks(nBlocks, nThreads, heliostatArgument->numberOfMicroHeliostats * sunrayArgument.numberOfLightsPerGroup, true);

    map_raytracing << < nBlocks, nThreads >> >
                                 (sunrayArgument, *rectangleReceiver, *rectangleGrid, heliostatArgument, d_subHeliostat_vertexes, factor);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}
