#include "hip/hip_runtime.h"
//
// Created by feng on 19-3-29.
// PS: Define the rectangle grid data structure.
//
#include "RectangleGrid.cuh"
#include "global_function.cuh"

void RectangleGrid::Cinit(){
    subgrid_num_.x = int(ceil(size_.x / interval_.x));
    subgrid_num_.y = int(ceil(size_.y / interval_.y));
    subgrid_num_.z = int(ceil(size_.z / interval_.z));
}

void RectangleGrid::CClear(){
    if(d_grid_helio_index_){
        checkCudaErrors(hipFree(d_grid_helio_index_));
        d_grid_helio_index_ = nullptr;
    }

    if(d_grid_helio_match_){
        checkCudaErrors(hipFree(d_grid_helio_match_));
        d_grid_helio_match_ = nullptr;
    }
}


int boxIntersect(int helioBeginId, int subHelioSize, float3 min_pos, float3 max_pos,
        const RectangleGrid &grid, vector<vector<int> > &grid_helio_match_vector){
//    cout << "【helioBeginID:】 "<< helioBeginId << endl;
    int size = 0;
    float3 pos = grid.getPosition();
    float3 interval = grid.getInterval();
    int3 subgrid_num = grid.getSubGridNumber();

    int3 min_grid_pos = make_int3((min_pos - pos) / interval);
    int3 max_grid_pos = make_int3((max_pos - pos) / interval);

    if(min_grid_pos.x < 0 || min_grid_pos.y < 0 || min_grid_pos.z < 0
        || max_grid_pos.x >= subgrid_num.x || max_grid_pos.y >= subgrid_num.y || max_grid_pos.z >= subgrid_num.z){
        std::cerr << "The heliostats may out of the grid boundary. Please check your input file." << std::endl;
    }

    for(int x = max(0 , min_grid_pos.x); x <= min(subgrid_num.x - 1 , max_grid_pos.x); ++x){
        for(int y = max(0, min_grid_pos.y); y <= min(subgrid_num.y - 1, max_grid_pos.y); ++y){
            for(int z = max(0, min_grid_pos.z); z <= min(subgrid_num.z - 1, max_grid_pos.z); ++z){
                int pos_id = (x * subgrid_num.y * subgrid_num.z)
                                + (y * subgrid_num.z)
                                + z;
                for(int i = 0; i < subHelioSize; ++i){
                    grid_helio_match_vector[pos_id].push_back(helioBeginId + i);
                }
                size += subHelioSize;
            }
        }
    }
//    cout << "----------boxIntersect return size: ------------" << size << endl;
    return size;
}


//Set *d_grid_helio_match_, *d_grid_helio_index_ and num_grid_helio_match_
int RectangleGrid::CGridHelioMatch(const vector<Heliostat *> &h_helios){
    if(d_grid_helio_match_ || d_grid_helio_index_){
        throw std::runtime_error("The grid and heliostats corresponding relationship should be empty before calling this method.");
    }

    int start_subhelio_pos = 0;
    float3 minPos, maxPos;
    float radius = 0.0f;
    num_grid_helio_match_ = 0;

    vector<vector<int> > grid_helio_match_vector(subgrid_num_.x * subgrid_num_.y * subgrid_num_.z, vector<int>());
    //cout << "start_helio_index_: " << start_helio_index_ << endl;
    //cout << "num_helios_: " << num_helios_ << endl;
    for(int i = start_helio_index_; i < start_helio_index_ + num_helios_; ++i){
        float3 pos = h_helios[i]->getPosition();
        radius = length(h_helios[i]->getSize())/2;

        minPos = pos - radius;
        maxPos = pos + radius;
        //cout << "No." << i << " helio's minPos: " << minPos.x << ", " << minPos.y << ", " << minPos.z << endl;
        //cout << "No." << i << " helio's maxPos: " << maxPos.x << ", " << maxPos.y << ", " << maxPos.z << endl;

        num_grid_helio_match_ += boxIntersect(start_subhelio_pos, h_helios[i]->getSubHelioSize(), minPos, maxPos, *this, grid_helio_match_vector);
        start_subhelio_pos += h_helios[i]->getSubHelioSize();
    }

    int *h_grid_helio_index = new int[subgrid_num_.x * subgrid_num_.y * subgrid_num_.z + 1];
    h_grid_helio_index[0] = 0;

    int *h_grid_helio_match = new int[num_grid_helio_match_];

    int index = 0;
    for(int i = 0; i < subgrid_num_.x * subgrid_num_.y * subgrid_num_.z; ++i){
//        cout << "grid_helio_match_vector[i].size():" << grid_helio_match_vector[i].size() << endl;
//        for(int p = 0; p < grid_helio_match_vector[i].size(); p++){
//            cout << grid_helio_match_vector[i][p] << " ";
//        }
//        cout << endl;
        h_grid_helio_index[i + 1] = int(h_grid_helio_index[i] + grid_helio_match_vector[i].size());
//        cout << "h_grid_helio_index[" << i+1 << "] = " << h_grid_helio_index[i] << " + " << grid_helio_match_vector[i].size() << " = " << h_grid_helio_index[i+1] << endl;
        for(int j = 0; j < grid_helio_match_vector[i].size(); ++j, ++index){
            h_grid_helio_match[index] = grid_helio_match_vector[i][j];
        }
    }

    global_func::cpu2gpu(d_grid_helio_index_, h_grid_helio_index, subgrid_num_.x * subgrid_num_.y * subgrid_num_.z + 1);
    global_func::cpu2gpu(d_grid_helio_match_, h_grid_helio_match, num_grid_helio_match_);

    delete[] h_grid_helio_index;
    delete[] h_grid_helio_match;

    h_grid_helio_index = nullptr;
    h_grid_helio_match = nullptr;

    return start_subhelio_pos;
}


/**
 * Getter and setter of attributes for Rectangle Grid
 */
void RectangleGrid::setGridNumber(int3 grid_num){
    subgrid_num_ = grid_num;
}

void RectangleGrid::setDeviceGridHeliostatMatch(int *d_grid_helio_match){
    d_grid_helio_match_ = d_grid_helio_match;
}

void RectangleGrid::setDeviceGridHeliostatIndex(int *d_grid_helio_index){
    d_grid_helio_index_ = d_grid_helio_index;
}

size_t RectangleGrid::getNumberOfGridHeliostatMatch() const{
    return num_grid_helio_match_;
}

void RectangleGrid::setNumberOfGridHeliostatMatch(size_t num_grid_helio_match){
    num_grid_helio_match_ = num_grid_helio_match;
}
