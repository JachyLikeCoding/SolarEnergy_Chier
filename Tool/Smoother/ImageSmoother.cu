#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include "ImageSmoother.cuh"
#include <stdio.h>
#include <iostream>

__device__ __host__ bool insert2(float *element_entry, int pos, float elem){
    element_entry[pos] = elem;
    return true;
}


// Simple transformation kernel
__global__ void trimmed_mean(float *output, hipTextureObject_t texObj,
                             int kernel_radius, float trimmed_ratio, int width, int height){
    // Step 1: Calculate normalized texture coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x > width - 1 || y > height - 1){    // Out of range
        return;
    }

    extern __shared__ float sdata[];
    int k = (2 * kernel_radius + 1) * (2 * kernel_radius + 1) * trimmed_ratio + 1;
    //printf("k = %d\n", k);
    const int myId = threadIdx.x + threadIdx.y * blockDim.x;

    float *max_heap_entry = &sdata[2 * myId * k];
    float *min_heap_entry = (float *)&sdata[(2 * myId + 1) * k];

    int2 left_up = make_int2((x - kernel_radius >= 0) ? (x - kernel_radius) : 0,
                             (y - kernel_radius >= 0) ? (y - kernel_radius) : 0);

    int2 right_bottom = make_int2((x + kernel_radius < width) ? (x + kernel_radius) : width - 1,
                                  (y + kernel_radius < height) ? (y + kernel_radius) : height - 1);

    int cnt = (right_bottom.x - left_up.x + 1) * (right_bottom.y - left_up.y + 1);
    //printf("\tcnt = %d\n", cnt);

    k = cnt * trimmed_ratio;
    //printf("k = %d\n", k);

    Max_heap max_heap(k, max_heap_entry);
    Min_heap min_heap(k, min_heap_entry);
    float cur_value;
    float value = 0.0f;

    for(int r = left_up.y; r <= right_bottom.y; ++r){
        for(int c = left_up.x; c <= right_bottom.x; ++c){
            cur_value = tex2D<float>(texObj, float(c) + 0.5, float(r) + 0.5);
            value += cur_value;
            if(!max_heap.full()){
                max_heap.insert(cur_value);
                min_heap.insert(cur_value);
            }else if(cur_value < max_heap.top()){
                max_heap.delete_and_insert(cur_value);
            }else if(cur_value > min_heap.top()){
                min_heap.delete_and_insert(cur_value);
            }
        }

        // Step 2: Read from texture amd write to global memory
        output[y * width + x] = (value - max_heap.sum() - min_heap.sum()) / float(cnt - 2 * k);
    }
}



void ImageSmoother::image_smooth(float *d_array, int kernel_radius, float trimmed_ratio, int width, int height) {
    size_t size = width * height * sizeof(float);

    // Step 1: Allocate CUDA array in device memory
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &floatTex, width, height);

    // Step 2: Copy to device memory
    hipMemcpyToArray(cuArray, 0, 0, d_array, size, hipMemcpyDeviceToDevice);

    // Step 3: Define hipResourceDesc
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Step 4: Define hipTextureDesc
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;

    // Step 5: Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    // Step 6: Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    // int kernel_radius = 4;
    // float trimmed_ratio = 0.05;
    int heap_size = (2 * kernel_radius + 1) * (2 * kernel_radius + 1) * trimmed_ratio + 1;
    //printf("\nheap_size = %d\n", heap_size);
    trimmed_mean << <dimGrid, dimBlock, sizeof(float) * heap_size * 2 * dimBlock.x * dimBlock.y >> >
                                        (d_array, texObj, kernel_radius, trimmed_ratio, width, height);

    // Finally, clean up
    //  1) Destroy texture object
    hipDestroyTextureObject(texObj);
    //  2) Free device memory
    hipFreeArray(cuArray);
}